#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\functional.h>

using namespace std;
using namespace thrust;

#include "hip/hip_runtime.h"
#include ""

// one side of the matrix
#define MAX_SIZE 8192
// the width of the matrix
#define SIZE (MAX_SIZE * MAX_SIZE)


// make a template struct for kernel function parameter
template <typename T>
struct KernelArray {
	T* _array;
	int _size;
};


// kernel funciton can't pass by device_vector for each parameters
// than we should have to convert struct or pointer value
template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec) {
	KernelArray<T> kArray;
	kArray._array = thrust::raw_pointer_cast(&dVec[0]);
	kArray._size = (int)dVec.size();

	return kArray;
}

// kernel function for matrix multiplication
__global__ void mat_mul_kernel(KernelArray<int> dv1, KernelArray<int> dv2, KernelArray<int> tmp) {
	int col, row;
	int res = 0;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	for (int i = 0; i < MAX_SIZE; i++) {
		res += dv1._array[MAX_SIZE*col + i] * dv2._array[MAX_SIZE*i + row];
	}
	tmp._array[col + MAX_SIZE*row] = res;
}

// function for matrix multiplication
// initialize & call kernel function : mat_mul_kernel()
float mat_mul(host_vector<int> v1, host_vector<int> v2, host_vector<int> v3) {
	// device vector variable for calculating 
	// matrix multiplication in GPU
	device_vector<int> dv1(SIZE);
	device_vector<int> dv2(SIZE);
	device_vector<int> tmp(SIZE);

	// variable for checking runtime
	hipEvent_t start, stop;
	float t = 0.0;

	// initialize device variable, using host_vector
	dv1 = v1;
	dv2 = v2;
	// divide grid to block  && block to thread
	dim3 dimBlock(16, 16);
	dim3 dimGrid(16, 16);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// in kernel function, parameters cannot be a device_vector variable
	// we must convert to it's address
	mat_mul_kernel << <dimGrid, dimBlock >> > (convertToKernel(dv1), convertToKernel(dv2), convertToKernel(tmp));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// restore result device to host variable
	v3 = tmp;

	// return running time
	return t;
}



int main(int argc, char **argv) {
	host_vector<int> v1(SIZE, 1);
	host_vector<int> v2(SIZE, 1);
	host_vector<int> temp(SIZE);
	cout << "CUDA PROGRAM" << endl;
	cout << MAX_SIZE << " x " << MAX_SIZE << "matrix multiplication > " << mat_mul(v1, v2, temp) << endl;

	return 0;
}